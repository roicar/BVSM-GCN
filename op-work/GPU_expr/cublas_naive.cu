#include <iostream>
//#include <malloc.h>
#include <fstream>
#include <vector>
#include <sstream>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <filesystem>
#include <string>
#include <algorithm>
#include <hipblas.h>
#include <random>

#define N_RUNS 10
using namespace std;
namespace fs = std::filesystem;






/*
一个还不错的博客
https://blog.csdn.net/feng__shuai/article/details/105299959

#define cublasSgemm cublasSgemm_v2
CUBLASAPI cublasStatus_t CUBLASWINAPI cublasSgemm_v2
(
    cublasHandle_t handle,
    cublasOperation_t transa, cublasOperation_t transb,
    int m, int n, int k,
    const float *alpha,
    const float *A, int lda,
    const float *B, int ldb,
    const float *beta,
    float *C, int ldc
);

C = alpha*A*B + beta*C(看起来没啥问题，使用起来呵呵哒)
 
cublasHandle_t handle：调用 cuBLAS 库时的句柄
cublasOperation_t transa, 是否对A转置
cublasOperation_t transb, 是否对B转置
int m, int n, int k, mnk表示矩阵计算时候的维度
const float *alpha,
const float *A, //矩阵A
int lda,  //按列读取的长度
const float *B, 
int ldb, 
const float *beta,
float *C, 
int ldc ldc:按列取的个数

*/


//按行打印
template <typename T>
void print_matrix_row(T *data, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int l = i*n + j;
            cout << data[l] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

//获取特定开头的文件名
vector<string> getFileNames(const string& path, const string& prefix) {
    vector<string> fileNames;
    for (const auto& entry : fs::directory_iterator(path)) {
        if (entry.is_regular_file()) {
            string fileName = entry.path().filename().string();
            if (fileName.substr(0, prefix.length()) == prefix) {
                fileNames.push_back(fileName);
            }
        }
    }
    return fileNames;
}


//读取文件到指定容器内
vector<vector<float>> readFile(const string& filename) {
    ifstream file(filename);
    vector <vector<float>> data;

    if (file) {
        string line;
        while (std::getline(file, line)) {
            vector <float> row;
            stringstream lineStream(line);
            string cell;

            while (getline(lineStream, cell, ',')) {
                float cellf = stof(cell);
                row.push_back(cellf);
            }

            data.push_back(row);
        }

        file.close();
    } else {
        cout << "Failed to open file: " << filename << endl;
    }
    return data;
}



//将行读取的矩阵进行转置
void transpose(float *data,int m,int n){
    float *da;
    da= (float*)malloc(sizeof(float)*m*n);
    int count=0;
//    for(int i=0;i<n;i++){
//        for(int j=0;j<m;j++){
//            da[count]=data[j*n+i];
//            count++;
//        }
//    }
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            da[count]=data[j*n+i];
            count++;
        }
    }
    memcpy(data,da,sizeof(float)*m*n);
    free(da);
}

//验证实验结果
void verify(float* matrix1,float* matrix2,int m,int n){
    for(int i=0;i<m*n;i++){
        if(abs(matrix1[i]-matrix2[i])>1.0e-2){
            cout << "The result of matrix multiplication is wrong!" << endl;
            return;
        }

    }
    cout <<  "The result of matrix multiplication is true!" << endl;
}




float compute(const vector<vector<float>>& matrix_A, const vector<vector<float>>& matrix_B, const vector<vector<float>>& matrix_C)
{
    int m = matrix_A.size();
    int k = matrix_A[0].size();
    int n = matrix_B[0].size();
    cout<<"M: "<<matrix_A.size()<<endl;
    cout<<"K: "<<matrix_A[0].size()<<endl;
    cout<<"N: "<<matrix_B[0].size()<<endl;

//    int ldb=k;
//    int ldc=m;

    float *h_A,*h_B,*h_C,*hC_result;
    h_A=(float*)malloc(sizeof(float)*m*k);
    h_B=(float*)malloc(sizeof(float)*k*n);
    h_C=(float*)malloc(sizeof(float)*m*n);
    hC_result=(float*)malloc(sizeof(float)*m*n);

//    原数组数据计算
    cout<<"开始存储矩阵到一维数组"<<endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            h_A[i*k + j] = matrix_A[i][j];
        }
    }
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            h_B[i*n + j] = matrix_B[i][j];
        }
    }
    
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            hC_result[i*n + j] = matrix_C[i][j];
            //test
            //printf("%f  ",matrix_C[i][j]);
        }
        //cout<<endl;
    }
    cout<<"OK1"<<endl;

/*
    cout<<"随机数组准备生成"<<endl;
//  开始生成等维随机矩阵
    std::random_device rd;  // 随机设备
    std::mt19937 gen(rd()); // 使用随机设备生成随机种子
    std::uniform_real_distribution<float> dis(0.0, 1.0);  // 定义均匀分布的范围为（0，1）
    cout<<"随机种子已经完成"<<endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            matrix_A[i][j] = dis(gen);
            h_A[i*k + j] = matrix_A[i][j];
        }
    }
    cout<<"h_A没有问题"<<endl;
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            matrix_B[i][j] = dis(gen);
            h_B[i*n + j] = matrix_B[i][j];
        }
    }
    cout<<"矩阵C已经清空"<<endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrix_C[i][j] = 0;
            for(int t = 0; t < k; t++){
                matrix_C[i][j] += matrix_A[i][t] * matrix_B[t][j];
            }
            hC_result[i*n + j] = matrix_C[i][j];
        }
    }

    cout<<"随机数组已经生成完毕"<<endl;
*/

    //初始化h_C数组为全0
    memset(h_C,0, sizeof(float)*m*n);

    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 //transpose
    //transpose(h_B,k,n);
    //transpose(hC_result,m,n);

    // Allocate device memory
    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    float elapsedTime=0.0f;
    double time=0.0f;

    //warm-up
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // execute SpMM
    for(int run=0;run<N_RUNS;run++){
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    // Perform matrix multiplication

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    //protect real answer from test answer
    //cout<<"------------------------------------------------------------------"<<endl;
    /*
    for (int i = 0; i < m; i++) {
	for (int j = 0; j < n; j++) {
            ;
            printf("%f  ",h_C[i*n + j]);
            }
	cout<<endl;
	}
    */
    //printf("Verfication cusparse result: ");
          
    //verify(h_C,hC_result,m,n);

    time=elapsedTime/N_RUNS;

    float rel = time;


    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return rel;
}

int main(int argc, char *argv[]) {


    vector<vector<vector<float>>> matrix_A_list,matrix_B_list,matrix_C_list;

    float rel = 0.0;
    string DS;
    cin>>DS;
    
    string path = "/home/daiwen/model_Batched_GEMM/DataSets/R1/"+DS+"/result/";
    //string path = "/home/daiwen/model_Batched_GEMM/DataSets/"+DS+"/result/";
    //string path = "../DataSets/"+DS+"/result/";
    string prefix_A = "sparseA";
    string prefix_B = "denseB";
    string prefix_C = "denseC";
    vector<string> fileNames_A = getFileNames(path, prefix_A);
    vector<string> fileNames_B = getFileNames(path, prefix_B);
    vector<string> fileNames_C = getFileNames(path, prefix_C);
    sort(fileNames_A.begin(), fileNames_A.end());
    sort(fileNames_B.begin(), fileNames_B.end());
    sort(fileNames_C.begin(), fileNames_C.end());

    // 循环读取对应的矩阵放入列表
    for(auto filename_A : fileNames_A){
        matrix_A_list.push_back(readFile(path + filename_A));

    }

    for(auto filename_B : fileNames_B){
        matrix_B_list.push_back(readFile(path + filename_B));
    }

    for(auto filename_C : fileNames_C){
        matrix_C_list.push_back(readFile(path + filename_C));
    }
    int len = fileNames_A.size();
    cout<<"length of list_A:"<<len<<endl;

    for(int i = 0; i < len; i++){
        cout<<"The "<<i<<"th SpMM starts running"<<endl;
        float time = compute(matrix_A_list[i], matrix_B_list[i], matrix_C_list[i]);
        rel += time;
        cout<<"The compute cost is :"<<time<<endl;
        cout<<"The "<<i<<"th SpMM ends running"<<endl;
    }

    cout<<"total cost time is:"<<rel<<" ms";
    return 0;
}
